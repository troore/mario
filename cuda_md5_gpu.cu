// CUDA MD5 hash calculation implementation.

#define RSA_KERNEL md5_v2


#include <hip/hip_runtime.h>
#include <stdio.h>

typedef unsigned int uint;

//
// On-device variable declarations
//

extern __shared__ uint memory[];	// on-chip shared memory
__constant__ uint k[64], rconst[16];	// constants (in fast on-chip constant cache)
__constant__ uint target[4];		// target hash, if searching for hash matches

//
// MD5 magic numbers. These will be loaded into on-device "constant" memory
//
static const uint k_cpu[64] =
{
	0xd76aa478, 	0xe8c7b756,	0x242070db,	0xc1bdceee,
	0xf57c0faf,	0x4787c62a, 	0xa8304613,	0xfd469501,
	0x698098d8,	0x8b44f7af,	0xffff5bb1,	0x895cd7be,
	0x6b901122, 	0xfd987193, 	0xa679438e,	0x49b40821,

	0xf61e2562,	0xc040b340, 	0x265e5a51, 	0xe9b6c7aa,
	0xd62f105d,	0x2441453,	0xd8a1e681,	0xe7d3fbc8,
	0x21e1cde6,	0xc33707d6, 	0xf4d50d87, 	0x455a14ed,
	0xa9e3e905,	0xfcefa3f8, 	0x676f02d9, 	0x8d2a4c8a,

	0xfffa3942,	0x8771f681, 	0x6d9d6122, 	0xfde5380c,
	0xa4beea44, 	0x4bdecfa9, 	0xf6bb4b60, 	0xbebfbc70,
	0x289b7ec6, 	0xeaa127fa, 	0xd4ef3085,	0x4881d05,
	0xd9d4d039, 	0xe6db99e5, 	0x1fa27cf8, 	0xc4ac5665,

	0xf4292244, 	0x432aff97, 	0xab9423a7, 	0xfc93a039,
	0x655b59c3, 	0x8f0ccc92, 	0xffeff47d, 	0x85845dd1,
	0x6fa87e4f, 	0xfe2ce6e0, 	0xa3014314, 	0x4e0811a1,
	0xf7537e82, 	0xbd3af235, 	0x2ad7d2bb, 	0xeb86d391,
};

static const uint rconst_cpu[16] =
{
	7, 12, 17, 22,   5,  9, 14, 20,   4, 11, 16, 23,   6, 10, 15, 21
};

void init_constants()
{
	hipMemcpyToSymbol(HIP_SYMBOL(k), k_cpu, sizeof(k));
	hipMemcpyToSymbol(HIP_SYMBOL(rconst), rconst_cpu, sizeof(rconst));
}

//
// MD5 routines (straight from Wikipedia's MD5 pseudocode description)
//

__device__ inline uint leftrotate (uint x, uint c)
{
	return (x << c) | (x >> (32-c));
}

__device__ inline uint r(const uint i)
{
	return rconst[(i / 16) * 4 + i % 4];
}

__device__ inline uint &getw(uint *w, const int i)
{
	return w[(i+threadIdx.x) % 16];
}

__device__ inline uint getw(const uint *w, const int i)	// const- version
{
//	return w[(i+threadIdx.x) % 16];
	return w[i];
}


__device__ inline uint getk(const int i)
{
	return k[i];	// Note: this is as fast as possible (measured)
}

__device__ void step(const uint i, const uint f, const uint g, uint &a, uint &b, uint &c, uint &d, const uint *w)
{
	uint temp = d;
	d = c;
	c = b;
	b = b + leftrotate((a + f + getk(i) + getw(w, g)), r(i));
	a = temp;
}

__device__ void inline md5(const uint *w, uint &a, uint &b, uint &c, uint &d)
{
	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

	uint f, g, i = 0;
	for(; i != 16; i++)
	{
		f = (b & c) | ((~b) & d);
		g = i;
		step(i, f, g, a, b, c, d, w);
	}

	for(; i != 32; i++)
	{
		f = (d & b) | ((~d) & c);
		g = (5*i + 1) % 16;
		step(i, f, g, a, b, c, d, w);
	}

	for(; i != 48; i++)
	{
		f = b ^ c ^ d;
		g = (3*i + 5) % 16;
		step(i, f, g, a, b, c, d, w);
	}

	for(; i != 64; i++)
	{
		f = c ^ (b | (~d));
		g = (7*i) % 16;
		step(i, f, g, a, b, c, d, w);
	}

	a += a0;
	b += b0;
	c += c0;
	d += d0;
}

//////////////////////////////////////////////////////////////////////////////
/////////////       Ron Rivest's MD5 C Implementation       //////////////////
//////////////////////////////////////////////////////////////////////////////


/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z))) 

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }


/* Basic MD5 step. Transform buf based on in.
 */
void inline __device__ md5_v2(const uint *in, uint &a, uint &b, uint &c, uint &d)
{
	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

  /* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
  FF ( a, b, c, d, getw(in,  0), S11, 3614090360); /* 1 */
  FF ( d, a, b, c, getw(in,  1), S12, 3905402710); /* 2 */
  FF ( c, d, a, b, getw(in,  2), S13,  606105819); /* 3 */
  FF ( b, c, d, a, getw(in,  3), S14, 3250441966); /* 4 */
  FF ( a, b, c, d, getw(in,  4), S11, 4118548399); /* 5 */
  FF ( d, a, b, c, getw(in,  5), S12, 1200080426); /* 6 */
  FF ( c, d, a, b, getw(in,  6), S13, 2821735955); /* 7 */
  FF ( b, c, d, a, getw(in,  7), S14, 4249261313); /* 8 */
  FF ( a, b, c, d, getw(in,  8), S11, 1770035416); /* 9 */
  FF ( d, a, b, c, getw(in,  9), S12, 2336552879); /* 10 */
  FF ( c, d, a, b, getw(in, 10), S13, 4294925233); /* 11 */
  FF ( b, c, d, a, getw(in, 11), S14, 2304563134); /* 12 */
  FF ( a, b, c, d, getw(in, 12), S11, 1804603682); /* 13 */
  FF ( d, a, b, c, getw(in, 13), S12, 4254626195); /* 14 */
  FF ( c, d, a, b, getw(in, 14), S13, 2792965006); /* 15 */
  FF ( b, c, d, a, getw(in, 15), S14, 1236535329); /* 16 */
 
  /* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
  GG ( a, b, c, d, getw(in,  1), S21, 4129170786); /* 17 */
  GG ( d, a, b, c, getw(in,  6), S22, 3225465664); /* 18 */
  GG ( c, d, a, b, getw(in, 11), S23,  643717713); /* 19 */
  GG ( b, c, d, a, getw(in,  0), S24, 3921069994); /* 20 */
  GG ( a, b, c, d, getw(in,  5), S21, 3593408605); /* 21 */
  GG ( d, a, b, c, getw(in, 10), S22,   38016083); /* 22 */
  GG ( c, d, a, b, getw(in, 15), S23, 3634488961); /* 23 */
  GG ( b, c, d, a, getw(in,  4), S24, 3889429448); /* 24 */
  GG ( a, b, c, d, getw(in,  9), S21,  568446438); /* 25 */
  GG ( d, a, b, c, getw(in, 14), S22, 3275163606); /* 26 */
  GG ( c, d, a, b, getw(in,  3), S23, 4107603335); /* 27 */
  GG ( b, c, d, a, getw(in,  8), S24, 1163531501); /* 28 */
  GG ( a, b, c, d, getw(in, 13), S21, 2850285829); /* 29 */
  GG ( d, a, b, c, getw(in,  2), S22, 4243563512); /* 30 */
  GG ( c, d, a, b, getw(in,  7), S23, 1735328473); /* 31 */
  GG ( b, c, d, a, getw(in, 12), S24, 2368359562); /* 32 */

  /* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
  HH ( a, b, c, d, getw(in,  5), S31, 4294588738); /* 33 */
  HH ( d, a, b, c, getw(in,  8), S32, 2272392833); /* 34 */
  HH ( c, d, a, b, getw(in, 11), S33, 1839030562); /* 35 */
  HH ( b, c, d, a, getw(in, 14), S34, 4259657740); /* 36 */
  HH ( a, b, c, d, getw(in,  1), S31, 2763975236); /* 37 */
  HH ( d, a, b, c, getw(in,  4), S32, 1272893353); /* 38 */
  HH ( c, d, a, b, getw(in,  7), S33, 4139469664); /* 39 */
  HH ( b, c, d, a, getw(in, 10), S34, 3200236656); /* 40 */
  HH ( a, b, c, d, getw(in, 13), S31,  681279174); /* 41 */
  HH ( d, a, b, c, getw(in,  0), S32, 3936430074); /* 42 */
  HH ( c, d, a, b, getw(in,  3), S33, 3572445317); /* 43 */
  HH ( b, c, d, a, getw(in,  6), S34,   76029189); /* 44 */
  HH ( a, b, c, d, getw(in,  9), S31, 3654602809); /* 45 */
  HH ( d, a, b, c, getw(in, 12), S32, 3873151461); /* 46 */
  HH ( c, d, a, b, getw(in, 15), S33,  530742520); /* 47 */
  HH ( b, c, d, a, getw(in,  2), S34, 3299628645); /* 48 */

  /* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
  II ( a, b, c, d, getw(in,  0), S41, 4096336452); /* 49 */
  II ( d, a, b, c, getw(in,  7), S42, 1126891415); /* 50 */
  II ( c, d, a, b, getw(in, 14), S43, 2878612391); /* 51 */
  II ( b, c, d, a, getw(in,  5), S44, 4237533241); /* 52 */
  II ( a, b, c, d, getw(in, 12), S41, 1700485571); /* 53 */
  II ( d, a, b, c, getw(in,  3), S42, 2399980690); /* 54 */
  II ( c, d, a, b, getw(in, 10), S43, 4293915773); /* 55 */
  II ( b, c, d, a, getw(in,  1), S44, 2240044497); /* 56 */
  II ( a, b, c, d, getw(in,  8), S41, 1873313359); /* 57 */
  II ( d, a, b, c, getw(in, 15), S42, 4264355552); /* 58 */
  II ( c, d, a, b, getw(in,  6), S43, 2734768916); /* 59 */
  II ( b, c, d, a, getw(in, 13), S44, 1309151649); /* 60 */
  II ( a, b, c, d, getw(in,  4), S41, 4149444226); /* 61 */
  II ( d, a, b, c, getw(in, 11), S42, 3174756917); /* 62 */
  II ( c, d, a, b, getw(in,  2), S43,  718787259); /* 63 */
  II ( b, c, d, a, getw(in,  9), S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

}

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////


__host__ __device__ void md5_pad(char *paddedWord, char *gpuWord, uint len)
{
	uint i = 0;

	for (; i < len; i++)
		paddedWord[i] = gpuWord[i];
	paddedWord[i] = 0x80;

	i++;
	for (; i < 64; i++)
		paddedWord[i] = 0x0u;
	((uint *)paddedWord)[14] = len * 8; // bit length
}


// The kernel (this is the entrypoint of GPU code)
// Loads the 8-byte word to be hashed from global to shared memory
// and calls the calculation routine
__global__ void md5_calc(char *gpuWords, uint *gpuHashes, int realthreads, uint maxWordLen)
{
	uint idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x; // assuming blockDim.y = 1 and threadIdx.y = 0, always
	if (idx >= realthreads) { return; }

	// load the dictionary word for this thread
	uint *iPaddedWord = &memory[0] + threadIdx.x * 16;
	md5_pad ((char *)iPaddedWord, &gpuWords[maxWordLen * idx], maxWordLen);

	// compute MD5 hash
	uint a, b, c, d;

	RSA_KERNEL(iPaddedWord, a, b, c, d);

	// return the hash
	gpuHashes[4 * idx + 0] = a;
	gpuHashes[4 * idx + 1] = b;
	gpuHashes[4 * idx + 2] = c;
	gpuHashes[4 * idx + 3] = d;
}

// A helper to export the kernel call to C++ code not compiled with nvcc
double gpu_execute_kernel(int blocks_x, int blocks_y, int threads_per_block, int shared_mem_required, int realthreads, char *gpuWords, uint *gpuHashes, uint max_word_len)
{
	dim3 grid;
	grid.x = blocks_x; grid.y = blocks_y, grid.z = 1;

	hipEvent_t start, stop;
	hipEventCreate (&start), hipEventCreate (&stop);
	hipEventRecord (start, 0);

	md5_calc<<<grid, threads_per_block, shared_mem_required>>>(gpuWords, gpuHashes, realthreads, max_word_len);

	hipEventRecord (stop, 0);
	hipEventSynchronize (stop);
	float elapsedTime;
	hipEventElapsedTime (&elapsedTime, start, stop);
	hipEventDestroy (start), hipEventDestroy (stop);

	return elapsedTime;
}

